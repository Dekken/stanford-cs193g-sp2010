// This is machine problem 1, part 1, shift cypher
//
// The problem is to take in a string of unsigned chars and an int,
// the shift amount, and add the number to each element of
// the string, effectively "shifting" each element in the 
// string.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


// Repeating from the tutorial, just in case you haven't looked at it.

// "kernels" or __global__ functions are the entry points to code that executes on the GPU
// The keyword __global__ indicates to the compiler that this function is a GPU entry point.
// __global__ functions must return void, and may only be called or "launched" from code that
// executes on the CPU.

#define uchar unsigned char

void host_shift_cypher(uchar *input_array, uchar *output_array, int shift_amount, int alphabet_max, int array_length)
{
	int i;
	for(i=0;i<array_length;i++)
	{
		int element = input_array[i];
		int shifted = element + shift_amount;
		if(shifted > alphabet_max || shifted < 0)
		{
			shifted = shifted % (alphabet_max + 1);
		}
		output_array[i] = (uchar)shifted;
	}
}


// This kernel implements a per element shift
__global__ void shift_cypher(uchar *input_array, uchar *output_array, int shift_amount, int alphabet_max, int array_length)
{
	// your code here
}


int main(void)
{
  // create arrays of 256 elements
  int num_elements = 256;

  
  int alphabet_max = 255;
  
  // compute the size of the arrays in bytes
  int num_bytes = num_elements * sizeof(unsigned char);

  // pointers to host & device arrays
  uchar *host_input_array = 0;
  uchar *host_output_array = 0;
  uchar *host_output_checker_array = 0;
  uchar *device_input_array = 0;
  uchar *device_output_array = 0;
  

  // malloc a host array
  host_input_array = (uchar*)malloc(num_bytes);
  host_output_array = (uchar*)malloc(num_bytes);
  host_output_checker_array = (uchar*)malloc(num_bytes);

  // cudaMalloc two device arrays
  hipMalloc((void**)&device_input_array, num_bytes);
  hipMalloc((void**)&device_output_array, num_bytes);
  
  // if either memory allocation failed, report an error message
  if(host_input_array == 0 || host_output_array == 0 || host_output_checker_array == 0 || 
	device_input_array == 0 || device_output_array == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }

  // generate random input string
  // initialize
  srand(1);
  
  int shift_amount = rand();
  
  for(int i=0;i< num_elements;i++)
  {
	host_input_array[i] = (uchar)rand(); 
  }
  
  // copy input to GPU
  hipMemcpy(device_input_array, host_input_array, num_bytes, hipMemcpyHostToDevice);

  // choose a number of threads per block
  // 128 threads (4 warps) tends to be a good number
  int block_size = 128;

  int grid_size = num_elements / block_size;

  // launch kernel
  shift_cypher<<<grid_size,block_size>>>(device_input_array, device_output_array, shift_amount, alphabet_max, num_elements);

  // download and inspect the result on the host:
  hipMemcpy(host_output_array, device_output_array, num_bytes, hipMemcpyDeviceToHost);

  // generate reference output
  host_shift_cypher(host_input_array, host_output_checker_array, shift_amount, alphabet_max, num_elements);
  
  // check CUDA output versus reference output
  int error = 0;
  for(int i=0;i<num_elements;i++)
  {
	if(host_output_array[i] != host_output_checker_array[i]) 
	{ 
		error = 1;
	}
	
  }
  
  if(error)
  {
	printf("Output of CUDA version and normal version didn't match! \n");
  }
  else {
	printf("Worked! CUDA and reference output match. \n");
  }
 
  // deallocate memory
  free(host_input_array);
  free(host_output_array);
  free(host_output_checker_array);
  hipFree(device_input_array);
  hipFree(device_output_array);
}

